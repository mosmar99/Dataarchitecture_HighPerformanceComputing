#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>

// CUDA kernel for RGB to grayscale conversion
__global__ void rgb2grayKernel(uchar4* in_image, uchar4* out_image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        uchar4 pixel = in_image[index];

        // Calculate grayscale value
        unsigned char grayscale = 0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z;

        // Create a new uchar4 with the grayscale value for all components
        in_image[index] = make_uchar4(grayscale, grayscale, grayscale, 255); // Assuming 255 for alpha (fully opaque)
    }
}

int main(int argc, char** argv) {
    // create input/output streams
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output = videoOutput::Create(argc, argv, ARG_POSITION(1));
    videoOutput* output2 = videoOutput::Create(argc, argv, ARG_POSITION(2));

    uchar4* out_image = NULL;
    uint32_t N = input->GetWidth() * input->GetHeight();
    hipMalloc(&out_image, N*sizeof(uchar4));

    if (!input)
        return 0;

    // capture/display loop
    while (true) {
        uchar4* in_image = nullptr;
        int status = 0;

        if (!input->Capture(&in_image, 1000, &status)) {
            if (status == videoSource::TIMEOUT)
                continue;
            // 1000ms timeout (default)
            break; // EOS
        }

        if (output != nullptr) {
            // Define grid and block size for CUDA kernel launch
            dim3 blockDim(16, 16);
            dim3 gridDim((input->GetWidth() + blockDim.x - 1) / blockDim.x, (input->GetHeight() + blockDim.y - 1) / blockDim.y);

            // Launch the grayscale conversion kernel
            output->Render(in_image, input->GetWidth(), input->GetHeight());

            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(), input->GetHeight(), output->GetFrameRate());
            output->SetStatus(str);

            if (!output->IsStreaming()) // check if the user quit
                break;
        }

        if (output2 != nullptr) {
            // Define grid and block size for CUDA kernel launch
            dim3 blockDim(16, 16);
            dim3 gridDim((input->GetWidth() + blockDim.x - 1) / blockDim.x, (input->GetHeight() + blockDim.y - 1) / blockDim.y);

            // Launch the grayscale conversion kernel
            rgb2grayKernel<<<gridDim, blockDim>>>(in_image,out_image, input->GetWidth(), input->GetHeight());
            output2->Render(in_image, input->GetWidth(), input->GetHeight());

            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(), input->GetHeight(), output2->GetFrameRate());
            output2->SetStatus(str);

            if (!output2->IsStreaming()) // check if the user quit
                break;            
        }
    }

    return 0;
}
