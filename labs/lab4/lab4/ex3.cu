#include "hip/hip_runtime.h"
#include <jetson-utils/videoSource.h>
#include <jetson-utils/videoOutput.h>

typedef unsigned int hist_t;
// https://forums.developer.nvidia.com/t/gstreamer-gstdecoder-failed-to-retrieve-next-image-buffer/195168

__global__ void calcHistogramKernel(uchar4* gray_image, hist_t* histogram_device, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate index
    int index = threadIdx.y * blockDim.x + threadIdx.x;

    __shared__ hist_t histo_local[256]; 
    histo_local[index] = 0;
    __syncthreads();
 
    // calculate histogram_device
    if (x < width && y < height) {
        unsigned char gray = gray_image[y*width + x].x;
        atomicAdd(&(histo_local[gray]), 1);
    }
    __syncthreads();

    // write to global memory   
    atomicAdd(&(histogram_device[index]), histo_local[index]);
    __syncthreads();
}

__global__ void plotHistogramKernel(uchar4* image, unsigned int* histogram_device, int width, int height, int max_freq)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    uchar4 white_pixel = make_uchar4(255, 255, 255, 255);
    uchar4 black_pixel = make_uchar4(0, 0, 0, 255);
    unsigned char semi_black_val = 0;

    if (index < 256)
    {
        int freq = histogram_device[index] * 256 / max_freq;
        for (int i = 0; i < 256; i++)
        {
            int row = height - i - 1;
            semi_black_val = image[row * width + 2*index].x / 4;
            black_pixel = make_uchar4(semi_black_val, semi_black_val, semi_black_val, 255);
            if ( i <= freq) {
                image[row * width + 2*index]   = white_pixel;
                image[row * width + 2*index+1] = white_pixel;
            }            
            else
            {
                image[row * width + 2*index]   = black_pixel;
                image[row * width + 2*index+1] = black_pixel;
            }  
        }
    }
}

// CUDA kernel for RGB to grayscale conversion
__global__ void rgb2grayKernel(uchar4* in_image, uchar4* out_image, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;
        uchar4 pixel = in_image[index];

        // Calculate grayscale value
        unsigned char grayscale = 0.299f * pixel.x + 0.587f * pixel.y + 0.114f * pixel.z;

        // Create a new uchar4 with the grayscale value for all components
        out_image[index] = make_uchar4(grayscale, grayscale, grayscale, 255); // Assuming 255 for alpha (fully opaque)
    }
}

int main(int argc, char** argv) {
    // create input/output streams
    videoSource* input = videoSource::Create(argc, argv, ARG_POSITION(0));
    videoOutput* output2 = videoOutput::Create(argc, argv, ARG_POSITION(2));

    uchar4* out_image = NULL;
    uint32_t N = input->GetWidth() * input->GetHeight();
    hipMalloc(&out_image, N*sizeof(uchar4));

    hist_t *histogram_device = NULL;
    hipMalloc(&histogram_device, 256*sizeof(hist_t));
    hist_t histogram_host[256] = {0};

    unsigned int max_freq = 20000;

    if (!input)
        return 0;

    // capture/display loop
    while (true) {
        uchar4* in_image = nullptr;
        int status = 0;

        if (!input->Capture(&in_image, 1000, &status)) {
            if (status == videoSource::TIMEOUT)
                continue;
            // 1000ms timeout (default)
            break; // EOS
        }

        if (output2 != nullptr) {
            // Define grid and block size for CUDA kernel launch
            dim3 blockDim(16, 16);
            dim3 gridDim((input->GetWidth() + blockDim.x - 1) / blockDim.x, (input->GetHeight() + blockDim.y - 1) / blockDim.y);

            // Launch the grayscale conversion kernel
            rgb2grayKernel<<<gridDim, blockDim>>>(in_image, out_image, input->GetWidth(), input->GetHeight());
           
            // calculate histogram_device of grayscale image
            memset(histogram_host, 0, 256*sizeof(hist_t));
            hipMemcpy(histogram_device, histogram_host, 256*sizeof(hist_t), hipMemcpyHostToDevice);
            calcHistogramKernel<<<gridDim, blockDim>>>(out_image, histogram_device, input->GetWidth(), input->GetHeight());
            hipMemcpy(histogram_host, histogram_device, 256*sizeof(hist_t), hipMemcpyDeviceToHost);

            // // print histogram_host
            // for (int i = 0; i < 256; i++)
            // {
            //     printf("%d ", histogram_host[i]);
            // }
            // printf("\n");

            // plot histogram
            plotHistogramKernel<<<1, 256>>>(out_image, histogram_device, input->GetWidth(), input->GetHeight(), max_freq);

            // output final image 
            output2->Render(out_image, input->GetWidth(), input->GetHeight());

            // Update status bar
            char str[256];
            sprintf(str, "Camera Viewer (%ux%u) | %0.1f FPS", input->GetWidth(), input->GetHeight(), output2->GetFrameRate());
            output2->SetStatus(str);

            if (!output2->IsStreaming()) // check if the user quit
                break;            
        }
    }

    // destroy resources
    delete input;
    delete output2;

    return 0;
}