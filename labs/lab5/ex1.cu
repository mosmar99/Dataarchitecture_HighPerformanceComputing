#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Error checking macro
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void reduceSum(int* input, int* output, int n)
{
    extern __shared__ int partialSum[];
    unsigned int tid = threadIdx.x;
    unsigned int start = 2*blockIdx.x*blockDim.x;
    partialSum[tid] = input[start + tid];
    partialSum[blockDim.x+tid] = input[start + blockDim.x+tid];
    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
    {
        __syncthreads();
        if (tid % stride == 0)
            partialSum[2*tid] += partialSum[2*tid + stride];
    }
    __syncthreads();
    if (tid == 0)
        output[blockIdx.x] = partialSum[0];
}

int main(void) {
    const int numElements = 1 << 24;
    const int threadsPerBlock = 512;

    const int blocksPerGrid = (numElements + threadsPerBlock * 2 - 1) / (threadsPerBlock * 2);
    const int smemSize = 2 * threadsPerBlock * sizeof(int);
    int *h_input = (int *)malloc(numElements * sizeof(int));
    int *h_output = (int *)malloc(blocksPerGrid * sizeof(int));

    // Initialize the host input vector
    for (int i = 0; i < numElements; ++i) {
        h_input[i] = rand() % 100;
    }
    int *d_input, *d_output;
    cudaCheckError(hipMalloc((void **)&d_input, numElements * sizeof(int)));
    cudaCheckError(hipMalloc((void **)&d_output, blocksPerGrid * sizeof(int)));
    cudaCheckError(hipMemcpy(d_input, h_input, numElements * sizeof(int),
    hipMemcpyHostToDevice));

    // Launch the reduction kernel
    reduceSum<<<blocksPerGrid, threadsPerBlock, smemSize>>>(d_input, d_output, numElements);
    cudaCheckError(hipGetLastError());
    cudaCheckError(hipMemcpy(h_output, d_output, blocksPerGrid * sizeof(int),
    hipMemcpyDeviceToHost));

    // Complete the reduction on the CPU
    int totalSum = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        totalSum += h_output[i];
    }
    printf("Total Sum (GPU) = %d\n", totalSum);
    int totalSumCPU = 0;
    for (int i = 0; i < numElements; i++) {
        totalSumCPU += h_input[i];
    }
    printf("Total Sum (CPU) = %d\n", totalSumCPU);

    // Free device and host memory
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    return 0;
}