#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

// Error checking macro
#define cudaCheckError(ans) { cudaAssert((ans), __FILE__, __LINE__); }
inline void cudaAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"CUDA Error: %s at %s:%d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// __global__ void reduceSum(int* input, int* output, int n)
// {
//     extern __shared__ int partialSum[];
//     unsigned int tid = threadIdx.x;
//     unsigned int start = 2*blockIdx.x*blockDim.x;
//     partialSum[tid] = input[start + tid];
//     partialSum[blockDim.x+tid] = input[start + blockDim.x + tid];
//     for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2)
//     {
//         __syncthreads();
//         if (tid % stride == 0)
//             partialSum[2*tid] += partialSum[2*tid + stride];
//     }
//     __syncthreads();
//     if (tid == 0)
//         output[blockIdx.x] = partialSum[0];
// }

__global__ void reduceSum(int *input, int *output, int n) {
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load input into shared memory.
    extern __shared__ int sdata[];
    //sdata[tid] = input[tid] + input[tid + blockDim.x]
    sdata[tid] = (i < n) ? input[i] : 0; 
    __syncthreads();

#if 0
    // Perform reduction in shared memory.
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
#else
    if (tid < 256) sdata[tid] += sdata[tid + 256]; __syncthreads();
    if (tid < 128) sdata[tid] += sdata[tid + 128]; __syncthreads();
    if (tid < 64) sdata[tid] += sdata[tid + 64]; __syncthreads();
    if (tid < 32) sdata[tid] += sdata[tid + 32]; __syncthreads();
    if (tid < 16) sdata[tid] += sdata[tid + 16]; __syncthreads();
    if (tid < 8) sdata[tid] += sdata[tid + 8]; __syncthreads();
    if (tid < 4) sdata[tid] += sdata[tid + 4]; __syncthreads();
    if (tid < 2) sdata[tid] += sdata[tid + 2]; __syncthreads();
    if (tid < 1) sdata[tid] += sdata[tid + 1]; __syncthreads();
#endif
    // Write result for this block to output.
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

int main(void) {
    const int numElements = 1 << 24;
    const int threadsPerBlock = 512;

    const int blocksPerGrid = (numElements + threadsPerBlock - 1) / (threadsPerBlock);
    const int smemSize = threadsPerBlock * sizeof(int);
    int *h_input = (int *)malloc(numElements * sizeof(int));
    int *h_output = (int *)malloc(blocksPerGrid * sizeof(int));

    // Initialize the host input vector
    for (int i = 0; i < numElements; ++i) {
        h_input[i] = rand() % 100;
    }
    int *d_input, *d_output;
    cudaCheckError(hipMalloc((void **)&d_input, numElements * sizeof(int)));
    cudaCheckError(hipMalloc((void **)&d_output, blocksPerGrid * sizeof(int)));
    cudaCheckError(hipMemcpy(d_input, h_input, numElements * sizeof(int),
    hipMemcpyHostToDevice));

    // Launch the reduction kernel
    reduceSum<<<blocksPerGrid, threadsPerBlock, smemSize>>>(d_input, d_output, numElements);
    cudaCheckError(hipGetLastError());
    cudaCheckError(hipMemcpy(h_output, d_output, blocksPerGrid * sizeof(int),
    hipMemcpyDeviceToHost));

    // Complete the reduction on the CPU
    int totalSum = 0;
    for (int i = 0; i < blocksPerGrid; ++i) {
        totalSum += h_output[i];
    }
    printf("Total Sum (GPU) = %d\n", totalSum);
    int totalSumCPU = 0;
    for (int i = 0; i < numElements; i++) {
        totalSumCPU += h_input[i];
    }
    printf("Total Sum (CPU) = %d\n", totalSumCPU);

    // Free device and host memory
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    return 0;
}
